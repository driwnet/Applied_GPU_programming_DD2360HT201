
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

#define error 1e-6

#define NUM_ITERATIONS 1000
#define NUM_PARTICLES 10000
#define BLOCK_SIZE 256

struct particle {
    float position[3];
    float velocity[3];
};

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

__host__ __device__ void uptdateParticle(particle *particula, int iter, int id, int num_p){
    //update the velocity:
    particula[id].velocity[0] = (3*id + iter) % num_p;
    particula[id].velocity[1] = (4*id + iter) % num_p;
    particula[id].velocity[2] = (5*id + iter) % num_p;

    //update the position:
    particula[id].position[0] = particula[id].position[0] + particula[id].velocity[0]; 
    particula[id].position[1] = particula[id].position[1] + particula[id].velocity[1]; 
    particula[id].position[2] = particula[id].position[2] + particula[id].velocity[2]; 
}

__global__ void timeStep(particle *particles, int iter, int num_p){
    const int id = threadIdx.x + blockIdx.x*blockDim.x;
    if(id < num_p){
        uptdateParticle(particles, iter, id, num_p);
    }
}

void init_Array(particle *particulas){
    for(int i = 0; i < NUM_PARTICLES; i++){
        particulas[i].position[0] = rand() % 1000;
        particulas[i].position[1] = rand() % 1000;
        particulas[i].position[2] = rand() % 1000;

        particulas[i].velocity[0] = rand() % 1000;
        particulas[i].velocity[1] = rand() % 1000;
        particulas[i].velocity[2] = rand() % 1000;
    }
}

int main( int argc, char *argv[]){

    
    bool bien = true;
    int GRID = (NUM_PARTICLES + BLOCK_SIZE - 1)/BLOCK_SIZE;


    double start_GPU, stop_GPU;
    double start_CPU, stop_CPU;
    double diferencia_CPU, diferencia_GPU;



    particle *particlesCPU = (particle*)malloc(NUM_PARTICLES * sizeof(particle));
    particle *particlesGPU;
    particle *resCPU;

    hipHostMalloc(&resCPU, NUM_PARTICLES * sizeof(particle), hipHostMallocDefault);

    hipMalloc(&particlesGPU, NUM_PARTICLES * sizeof(particle));

    init_Array(particlesCPU);

    hipMemcpy(resCPU, particlesCPU, NUM_PARTICLES*sizeof(particle), hipMemcpyHostToHost);


    // CPU part//

    start_CPU = cpuSecond();

    for(int i = 0; i < NUM_ITERATIONS; i++){
        for(int j = 0; j < NUM_PARTICLES; j++){
            uptdateParticle(particlesCPU, i, j, NUM_PARTICLES);
        }
    }


    stop_CPU = cpuSecond();
    diferencia_CPU = stop_CPU - start_CPU;

    // Finish CPU part

    //Start GPU part

    start_GPU = cpuSecond();
    

    for(int i = 0; i < NUM_ITERATIONS; i++){
        

        hipMemcpy(particlesGPU, resCPU, sizeof(particle) * NUM_PARTICLES, hipMemcpyHostToDevice);

        
        timeStep<<<GRID, BLOCK_SIZE>>>(particlesGPU, i, NUM_PARTICLES);
        hipDeviceSynchronize();

        hipMemcpy(resCPU, particlesGPU, NUM_PARTICLES * sizeof(particle), hipMemcpyDeviceToHost);

    }

    stop_GPU = cpuSecond();

    diferencia_GPU = stop_GPU - start_GPU;

    for(int i = 0; i < NUM_PARTICLES && bien; i++){
        for(int dim = 0; dim < 3; dim++){
            if(fabs(particlesCPU[i].position[dim] - resCPU[i].position[dim]) > error ){
                bien = false;
                break;
            }
        }
    }

    printf("NUM_ITERATIONS: %d\n", NUM_ITERATIONS);
    printf("NUM_PARTICLES: %d\n", NUM_PARTICLES);
    printf("BLOCK_SIZE: %d\n", BLOCK_SIZE);
    if(bien){
        printf("datos correctos\n");
    }else{
        printf("datos incorrectos\n");
    }
        
    hipFree(particlesGPU);
    hipHostFree(resCPU);
    delete[] particlesCPU;
    //delete[] resCPU;

    printf("Duration of the CPU: %f\n", diferencia_CPU);
    printf("Duration of the GPU: %f\n", diferencia_GPU);
    printf("--------------------------------------------\n");
    
    return 0;
}
