
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

#define error 1e-6

#define NUM_ITERATIONS 1000
#define NUM_PARTICLES 10000
#define BLOCK_SIZE 256

struct particle {
    float position[3];
    float velocity[3];
};

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

__host__ __device__ void uptdateParticle(particle *particula, int iter, int id, int num_p){
    //update the velocity:
    particula[id].velocity[0] = (3*id + iter) % num_p;
    particula[id].velocity[1] = (4*id + iter) % num_p;
    particula[id].velocity[2] = (5*id + iter) % num_p;

    //update the position:
    particula[id].position[0] = particula[id].position[0] + particula[id].velocity[0]; 
    particula[id].position[1] = particula[id].position[1] + particula[id].velocity[1]; 
    particula[id].position[2] = particula[id].position[2] + particula[id].velocity[2]; 
}

__global__ void timeStep(particle *particles, int iter, int num_p){
    const int id = threadIdx.x + blockIdx.x*blockDim.x;
    if(id < num_p){
        uptdateParticle(particles, iter, id, num_p);
    }
}



int main( int argc, char *argv[]){

    
    bool bien = true;
    int GRID = (NUM_PARTICLES + BLOCK_SIZE - 1)/BLOCK_SIZE;


    double start_GPU, stop_GPU;
    double start_CPU, stop_CPU;
    double diferencia_CPU, diferencia_GPU;



    particle *particlesCPU;
    hipMallocManaged(&particlesCPU, NUM_PARTICLES * sizeof(particle));
    particle *particlesGPU;
    hipMallocManaged(&particlesGPU, NUM_PARTICLES * sizeof(particle));


    // CPU part//

    start_CPU = cpuSecond();

    for(int i = 0; i < NUM_ITERATIONS; i++){
        for(int j = 0; j < NUM_PARTICLES; j++){
            uptdateParticle(particlesCPU, i, j, NUM_PARTICLES);
        }
    }


    stop_CPU = cpuSecond();
    diferencia_CPU = stop_CPU - start_CPU;

    // Finish CPU part

    //Start GPU part

    start_GPU = cpuSecond();
    hipMalloc(&particlesGPU, NUM_PARTICLES * sizeof(particle));

    for(int i = 0; i < NUM_ITERATIONS; i++){
        timeStep<<<GRID, BLOCK_SIZE>>>(particlesGPU, i, NUM_PARTICLES);
        hipDeviceSynchronize();
    }

    

    stop_GPU = cpuSecond();

    diferencia_GPU = stop_GPU - start_GPU;

/*     for(int i = 0; i < NUM_PARTICLES && bien; i++){
        for(int dim = 0; dim < 3 && bien; dim++){
            if(abs(particlesCPU[i].position[dim] - particlesGPU[i].position[dim]) > error ){
                bien = false;
            }
        }
    } */

    printf("NUM_ITERATIONS: %d\n", NUM_ITERATIONS);
    printf("NUM_PARTICLES: %d\n", NUM_PARTICLES);
    printf("BLOCK_SIZE: %d\n", BLOCK_SIZE);
/*     if(bien){
        printf("datos correctos\n");
    }else{
        printf("datos incorrectos\n");
    } */
        
    hipFree(particlesGPU);
    hipFree(particlesCPU);

    printf("Duration of the CPU: %f\n", diferencia_CPU);
    printf("Duration of the GPU: %f\n", diferencia_GPU);
    printf("--------------------------------------------\n");
    
    return 0;
}