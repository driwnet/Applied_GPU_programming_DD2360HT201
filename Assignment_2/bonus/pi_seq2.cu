
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>
#include <sys/time.h>

#define SEED     921


#define TPB 256
#define NUM_ITER 1000000000
#define NUM_THREADS  10000
#define NUM_ITER_THREADS (NUM_ITER/NUM_THREADS)


double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}


__global__ void count_nom(int *d_res, hiprandState *states){
    const int idx = threadIdx.x + blockIdx.x*blockDim.x;
    double x,y,z;
    const int a = 1;

    if (idx >= NUM_THREADS) {
        return;
    }
    int seed = idx; // different seed per thread
    hiprand_init(seed, idx, 0, &states[idx]);


    for (int iter = 0; iter < NUM_ITER_THREADS; iter++) {
        x = hiprand_uniform (&states[idx]);
        y = hiprand_uniform (&states[idx]);
        z = sqrt((x*x) + (y*y));

        if (z <= 1.0)
        {
            atomicAdd(d_res, a);
        }
    }

}



int main(int argc, char* argv[])
{
    double pi;
    double start_time, stop_time, diference;
    const int grid = (NUM_THREADS + TPB - 1)/ TPB;
    
    int *d_res;
    int *count = (int*)malloc(sizeof(int));
    hipMalloc(&d_res, sizeof(int));

    srand(SEED); // Important: Multiply SEED by "rank" when you introduce MPI!
    
    hiprandState *dev_random;
    hipMalloc((void**)&dev_random, grid*TPB*sizeof(hiprandState));

    
    // Calculate PI following a Monte Carlo method
    start_time = cpuSecond();

    count_nom<<<grid, TPB>>>(d_res, dev_random);
    
    hipDeviceSynchronize();

    hipMemcpy(count, d_res,sizeof(int), hipMemcpyDeviceToHost);

    stop_time = cpuSecond();

    diference = stop_time - start_time;

    
    // Estimate Pi and display the result
    pi = ((double)count[0] / (double)(NUM_ITER_THREADS * NUM_THREADS)) * 4.0;
    

    printf("The result is %f\n", pi);
    printf("THREADS PER BLOCK: %d\n", TPB);
    printf("NUM_ITER_THREADS: %d\n", NUM_ITER_THREADS);
    printf("The execution time is %f\n", diference);
    
    return 0;
}